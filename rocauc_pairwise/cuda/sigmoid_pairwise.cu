
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void sigmoid_pairwise_loss_gpu(int size, float * loss, 
                                          const int * y_true, 
                                          const float * exp_pred)
{
    float eps = 1e-20;
    float P_hat = 0.;
    float P = 0.;
    int i = 0;

    float cur_loss = 0.;

    int tidx = threadIdx.x;
    int bidx = blockIdx.x;
    int bdimx = blockDim.x;

    i = (bidx*bdimx) + tidx;

    if(i < size){
        for(int j = i; j >= 0; j--){
            P_hat = 0.5 *(y_true[i] - y_true[j]) + 0.5;
            P = 1.0 / (1.0 + (exp_pred[j] / exp_pred[i]));
            cur_loss += P_hat*log(P + eps) + (1.0 - P_hat)*log(1.0 - P + eps);
        }
        atomicAdd(loss, cur_loss);
    }
}

__global__ void sigmoid_pairwise_grad_hess_gpu(int size, float * grad,
                                               float * hess,
                                               const int * y_true,
                                               const float * exp_pred)
{
    int tidx = threadIdx.x;
    int bidx = blockIdx.x;
    int bdimx = blockDim.x;

    int i = (bidx*bdimx) + tidx;

    float exp_tmp_diff = 0.;
    float cur_d_dx_i = 0.;
    float cur_d_dx_j = 0.;
    float cur_d2_dx2_i = 0.;
    float cur_d2_dx2_j = 0.;

    float P_hat = 0.;

    if(i < size)
    {
        for(int j = 0; j < i + 1; j++){
            P_hat = 0.5 *(y_true[i] - y_true[j]) + 0.5;
            exp_tmp_diff = exp_pred[i] / exp_pred[j];

            cur_d_dx_i = ((P_hat - 1.) * exp_tmp_diff + P_hat) / (exp_tmp_diff + 1.);
            cur_d_dx_j = -cur_d_dx_i;
            cur_d2_dx2_i = ((-exp_pred[i]*exp_pred[j]) / ((exp_pred[i] + exp_pred[j])*(exp_pred[i] + exp_pred[j])));
            cur_d2_dx2_j = cur_d2_dx2_i;
        
            atomicAdd(&(grad[i]), cur_d_dx_i);
            atomicAdd(&(grad[j]), cur_d_dx_j);
            atomicAdd(&(hess[i]), cur_d2_dx2_i);
            atomicAdd(&(hess[j]), cur_d2_dx2_j);
        }
    }
}